#include <iostream>
#include <hip/hip_runtime.h>

// CUDA核心函數: 計算陣列元素相加
__global__ void vecAdd(float* a, float* b, float* c, int n) {
    // 獲取全域索引
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // 確保索引在範圍內
    if (idx < n) {
        c[idx] = a[idx] + b[idx];
    }
}

int main() {
    const int N = 1000000;  // 陣列大小

    // 分配主機記憶體
    float* h_a = new float[N];
    float* h_b = new float[N];
    float* h_c = new float[N];

    // 初始化資料
    for (int i = 0; i < N; i++) {
        h_a[i] = static_cast<float>(i);
        h_b[i] = static_cast<float>(i * 2);
    }

    // 分配裝置記憶體
    float* d_a, * d_b, * d_c;
    hipMalloc(&d_a, N * sizeof(float));
    hipMalloc(&d_b, N * sizeof(float));
    hipMalloc(&d_c, N * sizeof(float));

    // 將資料從主機複製到裝置
    hipMemcpy(d_a, h_a, N * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, N * sizeof(float), hipMemcpyHostToDevice);

    // 設定區塊和執行緒配置
    const int blockSize = 256;
    const int numBlocks = (N + blockSize - 1) / blockSize;

    // 執行核心函數
    vecAdd<<<numBlocks, blockSize>>>(d_a, d_b, d_c, N);

    // 將結果從裝置複製到主機
    hipMemcpy(h_c, d_c, N * sizeof(float), hipMemcpyDeviceToHost);

    // 釋放裝置記憶體
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    // 驗證結果
    bool success = true;
    for (int i = 0; i < N; i++) {
        if (h_c[i] != h_a[i] + h_b[i]) {
            success = false;
            break;
        }
    }

    if (success) {
        std::cout << "Test passed!" << std::endl;
    } else {
        std::cout << "Test failed!" << std::endl;
    }

    // 釋放主機記憶體
    delete[] h_a;
    delete[] h_b;
    delete[] h_c;

    return 0;
}